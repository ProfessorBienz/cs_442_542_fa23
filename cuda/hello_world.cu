#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void print_kernel()
{
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main()
{
    print_kernel<<<10,1>>>();
    hipDeviceSynchronize();
    return 0;
}
