#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../timer.h"

__global__
void vecAddKernel(float* A, float* B, float* C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) C[i] = A[i] + B[i];
}

void vecAdd(float* A, float* B, float* C, int n)
{
    for (int i = 0; i < n; i++)
        C[i] = A[i] + B[i];
}

double sum(int n, float* h_C)
{
    double s = 0;
    for (int i = 0; i < n; i++)
        s += h_C[i];
    return s;
}

int main(int argc, char* argv[])
{
    float *h_A, *h_B, *h_C;
    double t0, tfinal;

    if (argc <= 1)
    {
        printf("Pass Vector Size as Command Line Arg\n");
        return 0;
    }

    int n = atoi(argv[1]);

    hipHostMalloc((void**)&h_A, n*sizeof(float));
    hipHostMalloc((void**)&h_B, n*sizeof(float));
    hipHostMalloc((void**)&h_C, n*sizeof(float));

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, n*sizeof(float));
    hipMalloc((void**)&d_B, n*sizeof(float));
    hipMalloc((void**)&d_C, n*sizeof(float));

    for (int i = 0; i < n; i++)
    {
        h_A[i] = 0.5;
        h_B[i] = 0.7;
    }

    t0 = get_time();
    vecAdd(h_A, h_B, h_C, n);
    tfinal = get_time() - t0;
    printf("VecAdd Time %e, Sum %e\n", tfinal, sum(n, h_C));


    // Copy host array to device array
    t0 = get_time();
    hipMemcpy(d_A, h_A, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n*sizeof(float), hipMemcpyHostToDevice);

    // Launch GPU Kernel
    vecAddKernel<<<ceil(n/256.0), 256>>>(d_A, d_B, d_C, n);

    hipMemcpy(h_C, d_C, n*sizeof(float), hipMemcpyDeviceToHost);
    tfinal = get_time() - t0;

    printf("VecAddKernel Time %e, Sum %e\n", tfinal, sum(n, h_C));

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return 0;
}
